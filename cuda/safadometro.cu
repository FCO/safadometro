
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

//função no device(VGA)
__global__ void safazeza(float *soma, float *dia, float *mes, float *ano, float *safadezaDevice){
		int tx = threadIdx.x;
		safadezaDevice[0] = soma[tx] + (ano[tx]/100)*(50-dia[tx]);
}

__global__ void anjo(float *safadezaTotal, float *anjoDevice){
	int tx = threadIdx.x;
	anjoDevice[0] = 100 - safadezaTotal[tx];
}

float somatorio(float mes){
	float soma = 0;

	while(mes > 0){
		soma += mes;
		mes--;
	}
	return soma;
}


//prepara as natrizes a serem enviadas para GPU
void preparaHost(float dia, float mes, float ano){
	int tam = 1;

	//host
	float soma = somatorio(mes);
	float *somaHost 			= 	(float *) malloc(tam * sizeof(float));
	float *diaHost  			= 	(float *) malloc(tam * sizeof(float));
	float *mesHost				=		(float *) malloc(tam * sizeof(float));
	float *anoHost				= 	(float *) malloc(tam * sizeof(float));
	float *safadezaHost 	=  	(float *) malloc(tam * sizeof(float));
	float *anjoHost				= 	(float *) malloc(tam * sizeof(float));

	somaHost[0] = soma;
	diaHost[0]	= dia;
	mesHost[0]	= mes;
	anoHost[0]	= ano;

	//Device
	float *somaDevice, *DiaDevice,*MesDevice,*anoDevice, *safadezaDevice;
	float *anjoDevice, *safadezaTotal;
	int size = tam * sizeof(int);


	hipMalloc((void **)&DiaDevice,size);
	hipMemcpy(DiaDevice,diaHost,size,hipMemcpyHostToDevice);

	hipMalloc((void **)&MesDevice,size);
	hipMemcpy(MesDevice,mesHost,size,hipMemcpyHostToDevice);

	hipMalloc((void **)&anoDevice,size);
	hipMemcpy(anoDevice,anoHost,size,hipMemcpyHostToDevice);

	hipMalloc((void **)&somaDevice,size);
	hipMemcpy(somaDevice,somaHost,size,hipMemcpyHostToDevice);

	hipMalloc((void **)&safadezaDevice,size); //Resultado safadeza
	safazeza<<<tam,tam>>>(somaDevice, DiaDevice, MesDevice, anoDevice, safadezaDevice);

	hipMemcpy(safadezaHost,safadezaDevice,size,hipMemcpyDeviceToHost);

	std::cout<<"Nivel de safazeda"<<std::endl;
	std::cout<<safadezaHost[0]<<std::endl;

	
	hipMalloc((void **)&anjoDevice,size); //Resultado anjo

	hipMalloc((void **)&safadezaTotal,size);
	hipMemcpy(safadezaTotal,safadezaHost,size,hipMemcpyHostToDevice);

	anjo<<<tam,tam>>>(safadezaTotal,anjoDevice);
	hipMemcpy(anjoHost,anjoDevice,size,hipMemcpyDeviceToHost);
	std::cout<<"Nivel de anjo"<<std::endl;
	std::cout<<anjoHost[0]<<std::endl;

}


int main(){
		float mes, ano, dia;

		std::cout<<"Digite o dia"<<std::endl;
		std::cin>>dia;

		std::cout<<"Digite o mes:"<<std::endl;
		std::cin>>mes;

		std::cout<<"Digite o ano "<<std::endl;
		std::cin>>ano;


		preparaHost(dia,mes,ano);

}
